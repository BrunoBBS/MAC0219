#include "hip/hip_runtime.h"
#include "acc.cuh"
#include "math.h"
#include "util.hpp"
#include <hip/hip_runtime.h>
#include <chrono>

using namespace std::chrono;

/* This code was copied from 
 * https://docs.nvidia.com/cuda/cuda-c-programming-guide/index.html#atomic-functions 
 * since there is no atomicAdd for GPUs with Compute Capability less than 6.0
 */
__device__ double atomicAdd(double* address, double val, double dummy)
{
    unsigned long long int* address_as_ull =
                              (unsigned long long int*)address;
    unsigned long long int old = *address_as_ull, assumed;

    do {
        assumed = old;
        old = atomicCAS(address_as_ull, assumed,
                        __double_as_longlong(val +
                               __longlong_as_double(assumed)));

    // Note: uses integer comparison to avoid hang in case of NaN (since NaN != NaN)
    } while (assumed != old);
    return __longlong_as_double(old);
}

__host__ double gpu_probing(uint64_t n_ops, int64_t M, int64_t k) 
{

    high_resolution_clock::time_point t1 = high_resolution_clock::now();

    gpu_integration(n_ops, M, k);

    high_resolution_clock::time_point t2 = high_resolution_clock::now();
    duration<double> dur = duration_cast<duration<double>>(t2 - t1);
    double flops         = n_ops / dur.count();
    return flops;
}

__device__ double gpu_f(int64_t M, int64_t k, double x)
{
    return (sin((2 * M + 1) * M_PI * x) * cos(2 * M_PI * k * x)) /
           sin(M_PI * x);
}

/** Calculates f for n_ops points and puts the sum in the global memory
 */
__global__ void gpu_calc(hiprandState_t *states, double *sum, double *sum_2,
                         uint64_t n_ops_thread, int64_t M, int64_t k,
                         uint64_t n_threads, uint64_t n_leap_ops)
{
    uint64_t tid = blockIdx.x * blockDim.x + threadIdx.x;

    __shared__ double block_sum;
    __shared__ double block_sum_2;

    if (threadIdx.x == 0)
        block_sum = block_sum_2 = 0;

    __syncthreads();

    if (tid < n_threads)
    {
        double x, res;
        hiprandState_t local_state = states[tid];

        n_ops_thread += tid < n_leap_ops;

        for (uint64_t i = 0; i < n_ops_thread; i++)
        {
            x   = hiprand_uniform_double(&local_state) / 2;
            res = gpu_f(M, k, x);
            atomicAdd(&block_sum, res, 0);
            atomicAdd(&block_sum_2, res * res, 0);
        }
        __syncthreads();
        if (threadIdx.x == 0)
        {
            atomicAdd(sum, block_sum, 0);
            atomicAdd(sum_2, block_sum_2, 0);
        }
    }
}

__host__ std::vector<double> gpu_integration(uint64_t n_ops, int64_t M,
                                             int64_t k)
{
    int block_size = 1024;

    // Divide the work
    // The GPU has n_ops operations to do
    uint64_t n_threads    = min((((uint64_t)1 >> 32) - 1) * block_size, n_ops);
    uint64_t n_blocks     = n_threads / block_size + (n_threads % block_size > 0);
    uint64_t n_ops_thread = n_ops / n_threads;
    uint64_t n_leap_ops   = n_ops % n_threads;

    /*********************************
     * CUDA environment and device setup
     *********************************/
    // States for the random generator
    hiprandState_t *states;
    // Allocate space for the states
    hipMalloc((void **)&states, n_ops * sizeof(hiprandState));
    // Pointer for the device global sum and sum_2 variables
    double *sum;
    // Allocate both variables as an array to save code lines :)
    hipMalloc((void **)&sum, 2 * sizeof(double));
    // I want to pass two variables to gpu_calc and not an array
    double *sum_2 = sum + 1;
    // Zero the variables as they will be part of a summation
    hipMemset(sum, 0, 2 * sizeof(double));

    // Actually call the calc function
    gpu_calc<<<n_blocks, block_size>>>(states, sum, sum_2, n_ops_thread, M, k, n_threads, n_leap_ops);

    std::vector<double> result(2);
    hipMemcpy(result.data(), sum, 2 * sizeof(double), hipMemcpyDeviceToHost);
    
    hipFree(states);
    hipFree(sum);

    return result;
}